#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

#include "stopthepop_QGS/stopthepop_common.cuh"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	float2* rects,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;
	// Generate no key/value pair for invisible Gaussians

	// QGS' reminder:
	// Is it possible for a Gaussian to have radii > 0 while tiles_touched = 0? No, it’s not possible: if tiles_touched = 0, 
	// radii would not be assigned a value in preprocessCUDA. 
	// However, it is possible for the Gaussian to have radii = 0 while tiles_touched \neq 0 (The issue originates from 
	// rounding errors in the forced type conversion in getRect). In this case, the code would not enter this if branch, 
	// and tile would not be assigned a valid tile ID | depth. As a result, the tile ID becomes a random value, 
	// leading to an out-of-bounds error.

	// This is typically more common in 2DGS/QGS, as being surface representations makes it more likely for the bounding box 
	// radius to be zero. In contrast, for 3DGS, achieving radius = 0 requires the eigenvalues of the covariance matrix to be 
	// zero, which is a much stricter condition.
	if (radii[idx] > 0) {
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], rects[idx], rect_min, rect_max, grid);
		int tiles_touched = (rect_max.x - rect_min.x) * (rect_max.y - rect_min.y);
		int tiles_sort = (idx == 0) ? offsets[idx] : offsets[idx] - offsets[idx - 1];
		if (tiles_sort != tiles_touched){
			printf("tiles_touched:%d\n",tiles_touched);
			printf("tiles_sort:%d\n\n",tiles_sort);
		}
		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}


// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges, int total_tiles_num)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	uint32_t depth = static_cast<uint32_t>(key & 0xFFFFFFFF);

	// If this if branch is entered, it indicates that some Gaussian primitives have not been assigned a key, 
	// causing currtile to be a random value. This is usually due to a Gaussian with an excessively small scale, 
	// resulting in radii = 0.
	if (currtile >= total_tiles_num){
		printf("currtile_ID:%u, depth:%f\n", currtile, *(float*)&depth);
	}
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.rects2D, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.view2gaussian, P * 16, 128);
	obtain(chunk, geom.rscales_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.scales_sign, P, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::PointState CudaRasterizer::PointState::fromChunk(char*& chunk, size_t P)
{
	PointState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.points2D, P, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N * 6, 128); // T, dist1, dist2, distortion_before_normalized, curv1, curv2
	obtain(chunk, img.center_depth, N, 128);
	obtain(chunk, img.center_alphas, N, 128);
	obtain(chunk, img.n_contrib, N * 2, 128);
	obtain(chunk, img.ranges, N, 128);
	obtain(chunk, img.point_ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;

}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	float* aabb,
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float sigma,
	const float* rotations,
	const float* view2gaussian_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float* cam_intr,
	const float kernel_size,
	const float* subpixel_offset,
	const bool prefiltered,
	const bool return_depth,
	const bool return_normal,
	float* out_colors,
	int* n_touched,
	int* radii,
	bool debug)
{
	const float focal_x = cam_intr[0];
	const float focal_y = cam_intr[1];
	const float principal_x = cam_intr[2];
	const float principal_y = cam_intr[3];

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		aabb,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		sigma,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		colors_precomp,
		view2gaussian_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		principal_x, principal_y,
		kernel_size,
		radii,
		geomState.rects2D,
		geomState.means2D,
		geomState.depths,
		geomState.view2gaussian,
		geomState.rgb,
		geomState.rscales_opacity,
		geomState.scales_sign,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
#if TILE_SORTING
    FORWARD::duplicate(
	P, width, height, focal_x, focal_y, principal_x, principal_y, sigma,
	geomState.means2D,
	geomState.depths,
	geomState.rscales_opacity,
	(float3*)scales,
	geomState.scales_sign,
	geomState.view2gaussian,
	geomState.point_offsets,
	radii,
	geomState.rects2D,
	binningState.point_list_keys_unsorted,
	binningState.point_list_unsorted,
	tile_grid
);
#else
	
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		geomState.rects2D,
		tile_grid)
#endif

	
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges,
			tile_grid.x * tile_grid.y);
	CHECK_CUDA(, debug)
	// printf("ok2\n");
	
	//printf("in CudaRasterizer::Rasterizer::forward, P: %d num_rendered: %d geo_chunk_size: %d img_chunk_size: %d, binning_chunk_size: %d\n", P, num_rendered, chunk_size, img_chunk_size, binning_chunk_size);
	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	const float* view2gaussian = view2gaussian_precomp != nullptr ? view2gaussian_precomp : geomState.view2gaussian;
	// const float* view2gaussian = view2gaussian_precomp;
	CHECK_CUDA(FORWARD::render(
		P,
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		focal_x, focal_y,
		principal_x, principal_y,
		sigma,
		feature_ptr,
		view2gaussian,
		viewmatrix,
		(float3*)means3D,
		(float3*)scales,
		geomState.depths,
		geomState.rscales_opacity,
		geomState.scales_sign,
		return_depth,
		return_normal,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_colors,
		n_touched), debug)

	return num_rendered;
}


// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* view2gaussian_precomp,
	const float* scales,
	const float scale_modifier,
	const float sigma,
	const bool stop_z_gradient,
	const bool reciprocal_z,
	const float* rotations,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float* cam_intr,
	const float tan_fovx, float tan_fovy,
	const float kernel_size,
	const float* subpixel_offset,
	const int* radii,
	const float* out_colors,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	float* dL_dview2gaussian,
	const bool return_depth,
	const bool return_normal,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_x = cam_intr[0];
	const float focal_y = cam_intr[1];
	const float principal_x = cam_intr[2];
	const float principal_y = cam_intr[3];

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	
	const float* view2gaussian_ptr = (view2gaussian_precomp != nullptr) ? view2gaussian_precomp : geomState.view2gaussian;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		focal_x, focal_y,
		principal_x, principal_y,
		sigma,
		stop_z_gradient,
		background,
		geomState.rscales_opacity,
		geomState.scales_sign,
		color_ptr,
		view2gaussian_ptr,
		viewmatrix,
		(float3*)means3D,
		(float3*)scales,
		geomState.depths,
		imgState.accum_alpha,
		imgState.n_contrib,
		out_colors,
		dL_dpix,
		return_depth,
		return_normal,
		dL_dopacity,
		dL_dcolor,
		dL_dscale,
	    dL_dview2gaussian), debug)

	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		width, height,
		reciprocal_z,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		focal_x, focal_y,
		scale_modifier,
		view2gaussian_ptr,
		viewmatrix,
		projmatrix,
		kernel_size,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dview2gaussian,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		dL_dopacity), debug)
}
